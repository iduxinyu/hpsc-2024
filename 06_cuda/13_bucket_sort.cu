
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucketSort(int* key, int* bucket, int range){

    int id=blockIdx.x*blockDim.x+threadIdx.x;

    

    if(id<range){
	bucket[id]=0;
    }

    __syncthreads();
    atomicAdd(&bucket[key[id]],1);

    int offset0=0, offset1=0;
    for(int i=0;i<range;i++){

	offset0=offset1;    
	offset1+=bucket[i];
	
	if(id<offset1 && id>=offset0){
	key[id]=i;

	
	continue;
	
	}
    }

}

int main() {
  int n = 50;
  int range = 5;

  int* key;
  int* bucket;
  hipMallocManaged(&key,n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));

//  std::vector<int> key(n);
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");
/*
  std::vector<int> bucket(range); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  }
*/
  bucketSort<<<1,n>>>(key,bucket,range);
  hipDeviceSynchronize();
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
