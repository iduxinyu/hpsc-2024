

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <chrono>
#include <math.h>

using namespace std;
typedef vector<vector<float>> matrix;


__global__ void init(float *u, float *v,float *p,float *b,float *un, float *vn, float *pn)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	//int j=id/nx;
	//int i=id%nx;

	u[id]=0;
	v[id]=0;
	p[id]=0;
	b[id]=0;
	un[id]=0;
	vn[id]=0;
	pn[id]=0;
}

__global__ void comB(float *u, float *v,float *b, int nx, int ny, double rho, double dx, double dy, double dt)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	int j=id/nx;
	int i=id%nx;

	if(j>=1&&j<ny-1&&i>=1&&i<nx-1)
	{
		b[j*nx+i]=rho * (1 / dt *
					((u[j*nx+i + 1] - u[j*nx+i - 1]) / (2 * dx) + (v[(j + 1)*nx+i] - v[(j - 1)*nx+i]) / (2 * dy)) -
					pow((u[j*nx+i + 1] - u[j*nx+i - 1]) / (2 * dx), 2) - 2 * ((u[(j + 1)*nx+i] - u[(j - 1)*nx+i]) / (2 * dy) *
					(v[j*nx+i + 1] - v[j*nx+i - 1]) / (2 * dx)) - pow((v[(j + 1)*nx+i] - v[(j - 1)*nx+i]) / (2 * dy), 2));
	}

}

__global__ void comP(float *p, float *pn, float *b, int nx, int ny, double dx, double dy, double dt)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	int j=id/nx;
	int i=id%nx;

	if(j>=1&&j<ny-1&&i>=1&&i<nx-1)
	{
		p[id] = (dy * dy * (pn[j*nx+i + 1] + pn[j*nx+i - 1]) +
						dx * dx * (pn[(j + 1)*nx+i] + pn[(j - 1)*nx+i]) -
						b[j*nx+i] * pow(dx, 2) * pow(dy, 2))
						/ (2 * (pow(dx, 2) + pow(dy, 2)));
	}

}

//可以并行吗？？ 有依赖性吗???????
__global__ void boundryP(float *p, int ny, int nx)
{

	int id=blockIdx.x*blockDim.x+threadIdx.x;

	int j=id/nx;
	int i=id%nx;

	if(j>=0&&j<ny)
	{
		p[j*nx+nx - 1] = p[j*nx+nx - 2];
		p[j*nx] = p[j*nx+1];
	}

	if(i>=0&&i<nx)
	{
		p[i] = p[nx+i];
		p[(ny - 1)*nx+i] = 0;
	}
}

__global__ void copyP(float* p, float* pn, int ny, int nx)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	pn[id]=p[id];

}

__global__ void copyUV(float *u, float *v, float *un, float *vn, int ny, int nx)
{

	int id=blockIdx.x*blockDim.x+threadIdx.x;

	int j=id/nx;
	int i=id%nx;

	if(j>=0&&j<ny&&i>=0&&i<nx)
	{
		un[id]=u[id];
		vn[id]=v[id];
	}
	
}

__global__ void comUV(float *u,float *v, float *un,float *vn, float *p, double dx,double dy, double dt, double rho, double nu,int nx, int ny)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	int j=id/nx;
	int i=id%nx;

	if(j>=1&&j<ny-1&&i>=1&&i<nx-1)
	{
		u[id] = un[id] - un[id] * dt / dx * (un[id] - un[id - 1])
					- un[id] * dt / dy * (un[id] - un[(j - 1)*nx+i])
					- dt / (2 * rho * dx) * (p[(j + 1)*nx+i] - p[(j - 1)*nx+i])
					+ nu * dt / pow(dx, 2) * (un[id + 1] - 2 * un[id] + un[id - 1])
					+ nu * dt / pow(dy, 2) * (un[(j + 1)+i] - 2 * un[id] + un[(j - 1)*nx+i]);

		
		v[id] = vn[id] - vn[id] * dt / dx * (vn[id] - vn[id - 1])
					- vn[id] * dt / dy * (vn[id] - vn[(j - 1)*nx+i])
					- dt / (2 * rho * dx) * (p[(j + 1)*nx+i] - p[(j - 1)*nx+i])
					+ nu * dt / pow(dx, 2) * (vn[id + 1] - 2 * vn[id] + vn[id - 1])
					+ nu * dt / pow(dy, 2) * (vn[(j + 1)*nx+i] - 2 * vn[id] + vn[(j - 1)*nx+i]);
	}

}

__global__ void boundryUV(float *u, float *v, int nx, int ny)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	int j=id/nx;
	int i=id%nx;

	if(j>=0&&j<ny)
	{
		u[j*nx] = 0;
		u[j*nx+nx - 1] = 0;
		v[j*nx] = 0;
		v[j*nx+nx - 1] = 0;
	}

	
	if(i>=0&&i<nx)
	{
			u[i] = 0;
			u[(ny - 1)*nx+i] = 1;
			v[i] = 0;
			v[(ny - 1)*nx+i] = 0;
	}

	u[id]=id;
	v[id]=id;

}

int main() {
	int nx = 41;
	int ny = 41;
	int nt = 500;
	int nit = 50;
	double dx = 2. / (nx - 1);
	double dy = 2. / (ny - 1);
	double dt = .01;
	double rho = 1.;
	double nu = .02;

	float *u,*v,*p,*b,*un,*vn,*pn;

	hipMallocManaged(&u,ny*nx*sizeof(float));
	hipMallocManaged(&v,ny*nx*sizeof(float));
	hipMallocManaged(&p,ny*nx*sizeof(float));
	hipMallocManaged(&b,ny*nx*sizeof(float));
	hipMallocManaged(&un,ny*nx*sizeof(float));
	hipMallocManaged(&vn,ny*nx*sizeof(float));
	hipMallocManaged(&pn,ny*nx*sizeof(float));

	init<<<ny,nx>>>(u, v, p , b , un, vn, pn);
	hipDeviceSynchronize();

	ofstream ufile("u.dat");
	ofstream vfile("v.dat");
	ofstream pfile("p.dat");


	for (int n = 0; n < nt; n++) {
		
		comB<<<ny,nx>>>(u, v, b, nx, ny, rho, dx, dy, dt);
		hipDeviceSynchronize();


		for (int it = 0; it < nit; it++) {
			copyP<<<ny,nx>>>(p, pn, ny, nx);
			hipDeviceSynchronize();

			comP<<<ny,nx>>>(p, pn, b , nx, ny, dx, dy, dt);
			hipDeviceSynchronize();

			boundryP<<<ny,nx>>>(p, ny, nx);
			hipDeviceSynchronize();
			
		}
		
		copyUV<<<ny,nx>>>(u, v, un, vn, ny, nx);
		hipDeviceSynchronize();



		comUV<<<ny,nx>>>(u, v, un, vn,p, dx, dy, dt,  rho, nu, nx, ny);
		hipDeviceSynchronize();

		
		printf("n =%d \n", n);

		boundryUV<<<ny,nx>>>(u, v, nx, ny);
		hipDeviceSynchronize();

		if (n % 10 == 0) {
			for (int j = 0; j < ny; j++)
				for (int i = 0; i < nx; i++)
					ufile << u[j*nx+i] << " ";
			ufile << "\n";
			for (int j = 0; j < ny; j++)
				for (int i = 0; i < nx; i++)
					vfile << v[j*nx+i] << " ";
			vfile << "\n";
			for (int j = 0; j < ny; j++)
				for (int i = 0; i < nx; i++)
					pfile << p[j*nx+i] << " ";
			pfile << "\n";
		}
	}



	ufile.close();
	vfile.close();
	pfile.close();
}
